// use compute-sanitizer <executable name> to get more info
//bc of driver and nvcc version incompat, use nvcc -arch=sm_75 -gencode=arch=compute_75,code=sm_75 -o hello hello.cu to compile to to specific platform (titan)
#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void hello(){  
  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main(){

  hello<<<2, 2>>>(); // num threads divided by block size, block size
  hipDeviceSynchronize();
  return 0;
}

